#include "hip/hip_runtime.h"
#include "caffe/common.hpp"
#include "caffe/util/complex_functions.hpp"

namespace caffe {

__global__ void complex_kernel(const int n, const float* a, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = make_hipComplex(a[index], 0); }
}
__global__ void creal_kernel(const int n, const hipComplex* a, float* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCrealf(a[index]); }
}
__global__ void cimag_kernel(const int n, const hipComplex* a, float* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCimagf(a[index]); }
}
__global__ void cconj_kernel(const int n, const hipComplex* a, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipConjf(a[index]); }
}
__global__ void cadd_scalar_kernel(const int n, const hipComplex alpha, hipComplex* X) {
  CUDA_KERNEL_LOOP(index, n) { X[index] = hipCaddf(X[index], alpha); }
}
__global__ void cadd_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCaddf(a[index], b[index]); }
}
__global__ void csub_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCsubf(a[index], b[index]); }
}
__global__ void cmul_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCmulf(a[index], b[index]); }
}
__global__ void cdiv_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = hipCdivf(a[index], b[index]); }
}
void caffe_gpu_complex(const int N, const float* a, hipComplex* y) {
  complex_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, y);
}
void caffe_gpu_creal(const int N, const hipComplex* a, float* y) {
  creal_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, y);
}
void caffe_gpu_cimag(const int N, const hipComplex* a, float* y) {
  cimag_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, y);
}
void caffe_gpu_cconj(const int N, const hipComplex* a, hipComplex* y) {
  cconj_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, y);
}
void caffe_gpu_cadd_scalar(const int N, const hipComplex alpha, hipComplex* X) {
  cadd_scalar_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, X);
}
void caffe_gpu_cadd(const int N, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  cadd_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
}
void caffe_gpu_csub(const int N, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  csub_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
}
void caffe_gpu_cmul(const int N, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  cmul_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
}
void caffe_gpu_cdiv(const int N, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  cdiv_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b, y);
}

}
